#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:02:28 on Tue, Feb 28, 2023
//
// Description: mma naive hgemm

#include "common.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define WARP_SIZE 32

// see: https://zhuanlan.zhihu.com/p/650374808
__global__ void mmaNaiveKernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, size_t M,
                               size_t N, size_t K) {
    const size_t K_tiles = div_ceil(K, MMA_K);

    const size_t warp_row = blockIdx.y * MMA_M;
    const size_t warp_col = blockIdx.x * MMA_N;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    __shared__ half A_smem[MMA_M][MMA_K]; // 16 * 16 = 256 half, a warp have 32 threads, each threads handle 8 half = 4 float = 4 int = 1 int4
    __shared__ half B_smem[MMA_N][MMA_K]; // 8  * 16 = 128 half, a warp have 32 threads, each threads handle 4 half = 2 float = 2 int = 0.5 int4
    __shared__ half C_smem[MMA_M][MMA_N]; // 16 * 8  = 128 half, a warp have 32 threads, each threads handle 4 half = 2 float = 2 int = 0.5 int4

    const size_t lane_id = threadIdx.x % WARP_SIZE;

    uint32_t RC[2] = {0, 0};

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        *((int4 *)(&A_smem[lane_id / 2][0]) + lane_id % 2) =
            *((int4 *)(&A[(warp_row + lane_id / 2) * K + i * MMA_K]) + lane_id % 2);

        if (lane_id < MMA_N * 2) {
            *((int4 *)(&B_smem[lane_id / 2][0]) + lane_id % 2) =
                *((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) + lane_id % 2);
        }

        __syncthreads();

        // see https://docs.nvidia.com/cuda/parallel-thread-execution/#matrix-fragments-for-mma-m16n8k16-with-floating-point-type for register organization
        uint32_t RA[4];
        uint32_t RB[2];

        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_smem_lane_addr);

        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
        LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }
    // see: https://pica.zhimg.com/v2-aef0a60b6976aeaeaa922151232d7b8a_1440w.jpg
    *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
    *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

    __syncthreads();

    if (lane_id < MMA_M) {
        *((int4 *)(&C[(warp_row + lane_id) * N + warp_col])) = *((int4 *)(&C_smem[lane_id][0]));
    }
}

void mmaNaive(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

    mmaNaiveKernel<<<grid, block>>>(A, B, C, M, N, K);
}
